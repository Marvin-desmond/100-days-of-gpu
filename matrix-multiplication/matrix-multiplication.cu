#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define R1 5
#define C1 10
#define R2 10
#define C2 8

void initMatrix(int **A, int start_idx, int R, int C);
void flattenMatrix(int **M, int *M_flat, int R, int C);

void checkLastError() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA ERROR: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void matMulKernel(int *A, int *B, int *C, int M, int N, int P) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < M && col < P) {
        int pixel_value = 0;
        for (int i = 0; i < N; i++){
            pixel_value += A[row * N + i] * B[i * P + col];
        }
        C[row * P + col] = pixel_value;
    }
}

void matMul(int* A, int* B, int *C, int M, int N, int P) {
    int *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, M * N * sizeof(int));
    hipMalloc((void**)&B_d, N * P * sizeof(int));
    hipMalloc((void**)&C_d, M * P * sizeof(int));
    hipMemcpy(A_d, A, M*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*P*sizeof(int), hipMemcpyHostToDevice);
    // specifying grid size and block size
    dim3 dimGrid(1., 1., 1.);
    dim3 dimBlock(16., 16., 1.);
    matMulKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, M, N, P);
    hipDeviceSynchronize();
    hipMemcpy(C, C_d, M*P*sizeof(int), hipMemcpyDeviceToHost);
    checkLastError();
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(int argc, char **argv) {
   // dynamically allocate memory for matrix A 
   int **A = (int **)malloc(R1 * sizeof(int*));
   for (int i = 0; i < R1; i++){
     A[i] = (int*)malloc(C1*sizeof(int));
   }
   // dynamically allocate memory for matrix B
   int **B = (int**)malloc(R2*sizeof(int*));
   for (int i = 0; i < R2; i++){
    B[i] = (int*)malloc(C2*sizeof(int));
   }
   // initialize data as if we did arange then reshape in pytorch
   initMatrix(A, 0, R1, C1);
   initMatrix(B, R1 * C1 + 1, R2, C2);
   printf(" start for B: %d\n", R1 * C1 + 1);
   // Since CUDA expects a vector in a row-major format, 
   // let's flatten the input matrices to vector variables before sending the 
   // data to the GPU
   int *A_flat = (int*)malloc(R1*C1*sizeof(int));
   int *B_flat = (int*)malloc(R2*C2*sizeof(int));
   flattenMatrix(A, A_flat, R1, C1);
   flattenMatrix(B, B_flat, R2, C2);
   // declare the resulting matrix for holding the results
   int *C_flat = (int*)malloc(R1*C2*sizeof(int));
   matMul(A_flat, B_flat, C_flat, R1, C1, C2);
   // convert it back to matrix
   int **C = (int**)malloc(R1*sizeof(int*));
   for (int i = 0; i < R1; i++){
     C[i] = (int*)malloc(C2*sizeof(int));
   }
   // allocation to C matrix
   for (int i = 0; i < R1; i++){
    for (int j = 0; j < C2; j++){
        C[i][j] = C_flat[i * C2 + j];
    }
   }
   // let's print the values of the final matrix
   for (int i = 0; i < R1; i++){
    for (int j = 0; j < C2; j++){
        printf("%d, ", C[i][j]);
    }
    printf("\n");
   }

   free(A_flat);   
   free(B_flat);   
   free(C_flat);   
   free(A);
   free(B);
   free(C);
}

void initMatrix(int **A, int start_idx, int R, int C)
{
    for (int i = 0; i < R; i++){
        for (int j = 0; j < C; j++){
          A[i][j] = (i * C + j ) + start_idx;
        }
    }
}

void flattenMatrix(int **M, int *M_flat, int R, int C){
   for (int i = 0; i < R; i++){
      for (int j = 0; j < C; j++){
        M_flat[i * C + j] = M[i][j];
      }
   }
}
