#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define R1 5
#define C1 10
#define R2 10
#define C2 8

void initMatrix(int **A, int start_idx, int R, int C);
int main(int argc, char **argv) {
   // dynamically allocate memory for matrix A 
   int **A = (int **)malloc(R1 * sizeof(int*));
   for (int i = 0; i < C1; i++){
     A[i] = (int*)malloc(C1*sizeof(int));
   }
   // dynamically allocate memory for matrix B
   int **B = (int**)malloc(R2*sizeof(int*));
   for (int i = 0; i < C2; i++){
    B[i] = (int*)malloc(sizeof(int));
   }
   // initialize data as if we did arange then reshape in pytorch
   initMatrix(A, 0, R1, C1);
   initMatrix(B, R1 * C1 + 1, R1, C1);
   
   // Since CUDA expects a vector in a row-major format, 
   // let's flatten the input matrices before sending the 
   // data to the GPU
   free(A);
   free(B);
}

void initMatrix(int **A, int start_idx, int R, int C)
{
    for (int i = 0; i < R; i++){
        for (int j = 0; j < C; j++){
          A[i][j] = (i * C + j ) + start_idx;
        }
    }
}
